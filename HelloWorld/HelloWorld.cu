#include "hip/hip_runtime.h"
#ifndef CAFFE_COMMON_CUH_
#define CAFFE_COMMON_CUH_
#include <stdio.h>
#include "HelloWorld.h"

// __global__ functions, or "kernels", execute on the device
__global__ void hello_kernel()
{
  printf("Hello, world from the device!\n");
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  hello_kernel<<<1,1>>>();

  // wait for the device to finish so that we see the message
  hipDeviceSynchronize();

  return 0;
}

#endif