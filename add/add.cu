
#include <hip/hip_runtime.h>
#include <stdio.h>
// Kernel definition 
// Run on GPU
__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
	// Allocate & initialize host data - run on the host
    int a, b, c;         // host copies of a, b, c
    a = 2;
    b = 7;

    int *d_a, *d_b, *d_c; // device copies of a, b, c
    // Allocate space for device copies of a, b, c
    int size = sizeof(int);
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
	
    // Copy a & b from the host to the device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);
	
    // Copy result back to the host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	
    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    printf("answer:%d\n",c);
    return 0;
}